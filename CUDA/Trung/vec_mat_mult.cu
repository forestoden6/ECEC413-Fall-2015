#include "hip/hip_runtime.h"
/* Vector-matrix multiplication: Y = A * X.
 * Host code.
 * Author: Trung Do
 * Date: 11/15/2015
*/

#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <string.h>
#include <math.h>

// includes, project
//#include <hip/hip_runtime_api.h>

// includes, kernels
#include "vec_mat_mult_kernel.cu"

#define MIN_NUMBER 1
#define MAX_NUMBER 4

////////////////////////////////////////////////////////////////////////////////
// declarations, forward

extern "C"
void compute_gold(float*, const float*, const float*, unsigned int, unsigned int);

Matrix allocate_matrix_on_gpu(const Matrix M);
Matrix allocate_matrix(int num_rows, int num_columns, int init);
void copy_matrix_to_device(Matrix Mdevice, const Matrix Mhost);
void copy_matrix_from_device(Matrix Mhost, const Matrix Mdevice);
void vec_mat_mult_on_device_using_global_memory(const Matrix M, const Matrix N, Matrix P);
void vec_mat_mult_on_device_using_shared_memory(const Matrix M, const Matrix N, Matrix P);
void print_matrix(const Matrix M);
float get_random_number(int, int);
int checkResults(float *, float *, int, float);


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main(int argc, char** argv) {
	// Matrices for the program
	Matrix  A; // N x N matrix
	Matrix  X; // N x 1 vector
	Matrix  Y_cpu, Y_gpu_1, Y_gpu_2; // N x 1 vector
	
	// Initialize the random number generator with a seed value 
	srand(time(NULL));
	
	// Check command line arguments
	if(argc > 1){
		printf("Error. This program accepts no arguments. \n");
		exit(0);
	}		
	 
	// Allocate and initialize the matrices
	A  = allocate_matrix(MATRIX_SIZE, MATRIX_SIZE, 1); // Create a random N x N matrix
	X  = allocate_matrix(MATRIX_SIZE, 1, 1); // Create a random N x 1 vector 
	Y_cpu  = allocate_matrix(MATRIX_SIZE, 1, 0); // Allocate memory for the output vectors
	Y_gpu_1 = allocate_matrix(MATRIX_SIZE, 1, 0); 
	Y_gpu_2 = allocate_matrix(MATRIX_SIZE, 1, 0);
 
    	// compute the vector-matrix multiplication on the CPU for comparison    

	struct timeval start, stop;	
	gettimeofday(&start, NULL);	
	printf("Performing serial calculation using CPU. \n");
	
	compute_gold(Y_cpu.elements, A.elements, X.elements, A.num_rows, A.num_columns);
	
	gettimeofday(&stop, NULL);
	printf("Execution time = %fs. \n", (float)(stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float)1000000));

	
	// Perform the vector-matrix multiplication on the GPU using global memory
    	// Return the results in Y_gpu_1
	vec_mat_mult_on_device_using_global_memory(A, X, Y_gpu_1);
   
	// check if the device result is equivalent to the expected solution
    	printf("Checking against reference result. \n");
	int size_elements = NUM_ROWS;
	int res = checkResults(Y_cpu.elements, Y_gpu_1.elements, size_elements, 0.0001);
	printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");


   	 // Perform the vector-matrix multiplication on the GPU using global memory
    	// Return the results in Y_gpu_2
	vec_mat_mult_on_device_using_shared_memory(A, X, Y_gpu_2);
   
	// check if the device result is equivalent to the expected solution
   	 printf("Checking against reference result. \n");
    	res = checkResults(Y_cpu.elements, Y_gpu_2.elements, size_elements, 0.0001);
	printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");

	// Free host matrices
	free(A.elements); A.elements = NULL;
	free(X.elements); X.elements = NULL;
	free(Y_cpu.elements); Y_cpu.elements = NULL;
	free(Y_gpu_1.elements); Y_gpu_1.elements = NULL;
    free(Y_gpu_2.elements); Y_gpu_2.elements = NULL;

	return 0;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void vec_mat_mult_on_device_using_global_memory(const Matrix A, const Matrix X, Matrix Y){
	//Interface host call to the device kernel code and invoke the kernel	
	
	/* allocate space on gpu */
	Matrix gpu_a = allocate_matrix_on_gpu( A );
	Matrix gpu_x = allocate_matrix_on_gpu( X );
	Matrix gpu_y = allocate_matrix_on_gpu( Y );

	/* copy matrices to gpu  */
	copy_matrix_to_device( gpu_a, A );
	copy_matrix_to_device( gpu_x, X );

	/* set gpu parameters */
	dim3 dimBlock( 512, 1 );
	dim3 dimGrid( MATRIX_SIZE / dimBlock.x, 1 );

	struct timeval start, stop;	
	gettimeofday(&start, NULL);
	
	/* invoke kernel on gpu*/
	printf("Performing multiplication using global memory. \n");
	MatrixMulKernel_using_global_memory <<< dimGrid, dimBlock >>> (gpu_a.elements, gpu_x.elements, gpu_y.elements);

	/* end timer, print result */
	hipDeviceSynchronize();
	gettimeofday(&stop, NULL);
	printf("Execution time = %fs. \n", (float)(stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float)1000000));

	/* check for gpu errors */
	hipError_t err =  hipGetLastError();
	if ( hipSuccess != err ) {
		fprintf(stderr, " GPU kernel failed: %s.\n", hipGetErrorString(err));
	}

	/* copy result back to host */
	copy_matrix_from_device( Y, gpu_y );

	/* clean up mem on gpu */
	hipFree( gpu_a.elements );
	hipFree( gpu_x.elements );
	hipFree( gpu_y.elements );
}

void vec_mat_mult_on_device_using_shared_memory(const Matrix A, const Matrix X, Matrix Y){
	//Interface host call to the device kernel code and invoke the kernel	

	/* allocate space on gpu */
        Matrix gpu_a = allocate_matrix_on_gpu( A );
        Matrix gpu_x = allocate_matrix_on_gpu( X );
        Matrix gpu_y = allocate_matrix_on_gpu( Y );

        /* copy matrices to gpu  */
        copy_matrix_to_device( gpu_a, A );
        copy_matrix_to_device( gpu_x, X );

        /* set gpu parameters */
        dim3 dimBlock( 16, 16 );
        dim3 dimGrid( MATRIX_SIZE / dimBlock.x, 1 );

			struct timeval start, stop;	
	gettimeofday(&start, NULL);

		printf("Performing multiplication using shared memory. \n");
        /* invoke kernel on gpu*/
        MatrixMulKernel_using_shared_memory <<< dimGrid, dimBlock >>> (gpu_a.elements, gpu_x.elements, gpu_y.elements);

	/* end timer, print result */
	hipDeviceSynchronize();
	gettimeofday(&stop, NULL);
	printf("Execution time = %fs. \n", (float)(stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float)1000000));
        
	/* check for gpu errors */
        hipError_t err =  hipGetLastError();
        if ( hipSuccess != err ) {
                fprintf(stderr, " GPU kernel failed: %s.\n", hipGetErrorString(err));
        } 

        /* copy result back to host */
        copy_matrix_from_device( Y, gpu_y );

        /* clean up mem on gpu */
        hipFree( gpu_a.elements );
        hipFree( gpu_x.elements );
        hipFree( gpu_y.elements );
}


// Allocate a device matrix of same size as M.
Matrix allocate_matrix_on_gpu(const Matrix M){
    Matrix Mdevice = M;
    int size = M.num_rows * M.num_columns * sizeof(float);
    hipMalloc((void**)&Mdevice.elements, size);
    return Mdevice;
}

// Allocate a matrix of dimensions height*width
//	If init == 0, initialize to all zeroes.  
//	If init == 1, perform random initialization.
Matrix allocate_matrix(int num_rows, int num_columns, int init){
    	Matrix M;
    	M.num_columns = M.pitch = num_columns;
    	M.num_rows = num_rows;
    	int size = M.num_rows * M.num_columns;
		
	M.elements = (float*) malloc(size*sizeof(float));
	for(unsigned int i = 0; i < size; i++){
		if(init == 0) M.elements[i] = 0; 
		else
			M.elements[i] = get_random_number(MIN_NUMBER, MAX_NUMBER);
	}
    return M;
}	

// Copy a host matrix to a device matrix.
void copy_matrix_to_device(Matrix Mdevice, const Matrix Mhost)
{
    int size = Mhost.num_rows * Mhost.num_columns * sizeof(float);
    Mdevice.num_rows = Mhost.num_rows;
    Mdevice.num_columns = Mhost.num_columns;
    Mdevice.pitch = Mhost.pitch;
    hipMemcpy(Mdevice.elements, Mhost.elements, size, hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void copy_matrix_from_device(Matrix Mhost, const Matrix Mdevice){
    int size = Mdevice.num_rows * Mdevice.num_columns * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, hipMemcpyDeviceToHost);
}

// Prints the matrix out to screen
void print_matrix(const Matrix M){
	for(unsigned int i = 0; i < M.num_rows; i++){
		for(unsigned int j = 0; j < M.num_columns; j++)
			printf("%f ", M.elements[i*M.num_columns + j]);
		printf("\n");
	} 
	printf("\n");
}

// Returns a random floating-point number between the specified min and max values 
float get_random_number(int min, int max){
	return (float)floor((double)(min + (max - min + 1)*((float)rand()/(float)RAND_MAX)));
}

int 
checkResults(float *reference, float *gpu_result, int num_elements, float threshold)
{
    int checkMark = 1;
    float epsilon = 0.0;
    
    for(int i = 0; i < num_elements; i++)
        if(fabsf((reference[i] - gpu_result[i])/reference[i]) > threshold){
            checkMark = 0;
            break;
        }

    for(int i = 0; i < num_elements; i++)
        if(fabsf((reference[i] - gpu_result[i])/reference[i]) > epsilon){
            epsilon = fabsf((reference[i] - gpu_result[i])/reference[i]);
        }

    printf("Max epsilon = %f. \n", epsilon); 
    return checkMark;
}

