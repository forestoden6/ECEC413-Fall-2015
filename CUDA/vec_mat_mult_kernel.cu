#include "hip/hip_runtime.h"
/* Vector-Matrix multiplication: Y = A * X.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include "vec_mat_mult.h"

__global__ void vec_mat_kernel_naive(float *Ad, float *Xd, float *Yd)
{
	//Thread Index
	int threadX = threadIdx.x;
	//int threadY = threadIdx.y;
	
	//Block Index
	int blockX = blockIdx.x;
	//int blockY = blockIdx.y;
	
	//Find absolute position
	int col = blockDim.x * blockX + threadX;
	//int row = blockDim.y * blockY + threadY;
	
	double Y_temp = 0;
	for(int i = 0; i < MATRIX_SIZE; i++){
		double A_element = Ad[MATRIX_SIZE * col + i]; //Get all the values in the row of A
		double X_element = Xd[i]; //Get all values in X
		Y_temp += A_element * X_element;
	}
	
	Yd[col] = (float)Y_temp;
}


__global__ void vec_mat_kernel_optimized(float *Ad, float *Xd, float *Yd)
{
	__shared__ float A_tile[TILE_SIZE][TILE_SIZE];
	__shared__ float X_tile[TILE_SIZE];
	
	//Thread Index
	int threadX = threadIdx.x;
	int threadY = threadIdx.y;
	
	//Block Index
	int blockX = blockIdx.x;
	int blockY = blockIdx.y;
	
	//Find absolute position
	int col = blockDim.x * blockX + threadX;
	int row = blockDim.y * blockY + threadY;
	
	int i = 0;
	double Y_temp = 0;
	
	while(i < MATRIX_SIZE){
		if(i + threadX < MATRIX_SIZE && row < MATRIX_SIZE)
			A_tile[threadY][threadX] = Ad[row * MATRIX_SIZE + i + threadX];
		else
			A_tile[threadY][threadX] = 0.0f;
			
		if(threadX < 1 && col < MATRIX_SIZE)
			X_tile[threadY] = Xd[(i+threadY)*MATRIX_SIZE + col];
		/*else
			X_tile[threadY][threadX] = 0.0f;*/
			
		__syncthreads();
		
		if(threadX == 0)
			for(int temp = 0; temp < TILE_SIZE; temp++)
				Y_temp += A_tile[threadY][temp] * X_tile[temp];
			
		__syncthreads();
		
		i += TILE_SIZE;
	}

	if(col < 1 && row < MATRIX_SIZE)
		Yd[row] = (float)Y_temp;
}



#endif // #ifndef _MATRIXMUL_KERNEL_H_
